#include <iostream>
#include <hip/hip_runtime.h>

#define OFFSET(row,col,ld) ((row)*(ld) + (col))
// 
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&pointer)[0])

void cpu_sgemm(float *a,float *b,float *c,const int M,const int K,const int N)
{
    for(int m = 0; m < M; m++)
    {
        for(int n = 0; n < N; n++)
        {
            float fsum = 0.0;
            for(int k = 0; k < K; k++)
            {
                fsum += a[OFFSET(m,k,K)]*b[OFFSET(k,n,N)];
            }
            c[OFFSET(m,n,N)] = fsum;
        }
    }
}

__global__ void cuda_sgemm(float *a,float *b,float *c,const int M,const int K,const int N)
{
    const int n = blockIdx.x*blockDim.x + threadIdx.x;
    const int m = blockIdx.y*blockDim.y + threadIdx.y;

    if(m < M && n < N)
    {
        float fsum = 0.0;
        for(int k = 0; k < K; k++)
        {
            fsum += a[OFFSET(m,k,K)]*b[OFFSET(k,n,N)];
        }
        c[OFFSET(m,n,N)] = fsum;
    }
}

__global__ void cuda_sgemm2(float *a,float *b,float *c,const int M,const int K,const int N)
{
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty*blockDim.x + tx;

    const int BM = 128;
    const int BN = 128;
    const int TM = 8;
    const int TN = 8;
    const int BK = 8;

    __shared__ float s_a[BM][BK];
    __shared__ float s_b[BK][BN];

    float r_c[TM][TN];

    int load_a_smem_m = tid>>1;
    int load_a_smem_k = (tid&1)<<2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid&31)<<2;

    int load_a_gmem_m = by*BM + load_a_smem_m;
    int load_b_gmem_n = bx*BN + load_b_smem_n;

    for(int bk = 0; bk < (K+BK-1)/BK; bk++)
    {
        int load_a_gmem_k = bk*BK+load_a_smem_k;
        int load_b_gmem_k = bk*BK+load_b_smem_k;

        FLOAT4(s_a[load_a_smem_m][load_a_smem_k]) = FLOAT4(a[OFFSET(load_a_gmem_m,load_a_gmem_k,K)]);
        FLOAT4(s_b[load_b_smem_k][load_b_smem_n]) = FLOAT4(b[OFFSET(load_b_gmem_k,load_b_gmem_n,N)]);

        __syncthreads();

        for(int m = 0; m < TM; m++)
        {
            for(int n = 0; n < TN; n++)
            {
                int comp_a_m = ty*TM+m;
                int comp_b_n = tx*TN+n;
                for(int k = 0; k < BK; k++)
                {
                    r_c[m][n] +=s_a[comp_a_m][k]*s_b[k][comp_b_n];
                }
            }
        }
        __syncthreads();
    }

    for(int m = 0; m < TM; m++)
    {
        for(int n = 0; n< TN; n++)
        {
            int store_c_gmem_m = by*BM + tx*TM + m;
            int store_c_gmem_n = bx*BN + ty*TN + n;

            c[OFFSET(store_c_gmem_m,store_c_gmem_n,N)] = r_c[m][n];

        }
    }

}

int main(int argc,char* argv[])
{
    std::cout << "main:"<<std::endl;

    const int M = 64;
    const int K = 64;
    const int N = 64;

    float a[M*K] = {0};
    float b[K*N] = {0};
    float c[M*N] = {0};

    for(int i = 0; i < M*K; i++)
    {
        a[i] = 1.0;
    }

    for(int i = 0; i < K*N; i++)
    {
        b[i] = 1.0;
    }

    {
        //cpu_sgemm(a,b,c,M,K,N);    
    }
    {
        // float *a_d = nullptr;
        // float *b_d = nullptr;
        // float *c_d = nullptr;
        // cudaMalloc((void**)&a_d,sizeof(float)*M*K);
        // cudaMalloc((void**)&b_d,sizeof(float)*K*N);
        // cudaMalloc((void**)&c_d,sizeof(float)*M*N);
        
        // cudaMemcpy(a_d,a,sizeof(float)*M*K,cudaMemcpyHostToDevice);
        // cudaMemcpy(b_d,b,sizeof(float)*K*N,cudaMemcpyHostToDevice);

        // const int BM = 32;
        // const int BN = 32;
        // dim3 gridDim((N+BN -1)/BN,(M+BM-1)/BM);
        // dim3 blockDim(BN,BM);

        // cuda_sgemm<<<gridDim,blockDim,0>>>(a_d,b_d,c_d,M,K,N);

        // cudaMemcpy(c,c_d,sizeof(float)*M*N,cudaMemcpyDeviceToHost);
    }
    {
        float *a_d = nullptr;
        float *b_d = nullptr;
        float *c_d = nullptr;
        hipMalloc((void**)&a_d,sizeof(float)*M*K);
        hipMalloc((void**)&b_d,sizeof(float)*K*N);
        hipMalloc((void**)&c_d,sizeof(float)*M*N);
        
        hipMemcpy(a_d,a,sizeof(float)*M*K,hipMemcpyHostToDevice);
        hipMemcpy(b_d,b,sizeof(float)*K*N,hipMemcpyHostToDevice);

        const int BM = 128;
        const int BN = 128;
        const int TM = 8;
        const int TN = 8;

        dim3 gridDim((N+BN -1)/BN,(M+BM-1)/BM);
        dim3 blockDim(BN/TN,BM/TM);

        cuda_sgemm2<<<gridDim,blockDim,0>>>(a_d,b_d,c_d,M,K,N);

        hipMemcpy(c,c_d,sizeof(float)*M*N,hipMemcpyDeviceToHost);
    }

    for(int i = 0; i < 10; i++)
    {
        std::cout << c[i]<< std::endl;
    }

    return 0;
}
