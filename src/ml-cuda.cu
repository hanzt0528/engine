#include "ml-cuda.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <iostream>

static int g_device_count = -1;
static int g_main_device = 0;
static int g_compute_capabilities[GGML_CUDA_MAX_DEVICES];
static float g_tensor_split[GGML_CUDA_MAX_DEVICES] = {0};


#define MAX_STREAMS 8
static hipStream_t g_cudaStreams[GGML_CUDA_MAX_DEVICES][MAX_STREAMS] = { nullptr };

static bool g_cublas_loaded = false;
bool ggml_cublas_loaded(void) {
    return g_cublas_loaded;
}


static hipblasHandle_t g_cublas_handles[GGML_CUDA_MAX_DEVICES] = {nullptr};



/**
 * Panic wrapper for unwinding CUDA runtime errors
 */
#define CUDA_CHECK(status)                                              \
  {                                                                     \
    hipError_t error = status;                                         \
    if (error != hipSuccess) {                                         \
      std::cerr << "Got bad cuda status: " << hipGetErrorString(error) \
                << " at line: " << __LINE__ << std::endl;               \
      exit(EXIT_FAILURE);                                               \
    }                                                                   \
  }


#if CUDART_VERSION >= 12000
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            int id;                                                                     \
            hipGetDevice(&id);                                                         \
            fprintf(stderr, "\ncuBLAS error %d at %s:%d: %s\n",                         \
                    err_, __FILE__, __LINE__, cublasGetStatusString(err_));             \
            fprintf(stderr, "current device: %d\n", id);                                \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#else
#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            int id;                                                                     \
            hipGetDevice(&id);                                                         \
            fprintf(stderr, "\ncuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);  \
            fprintf(stderr, "current device: %d\n", id);                                \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)
#endif // CUDART_VERSION >= 11


inline hipError_t ggml_cuda_set_device(const int device) {
    int current_device;
    CUDA_CHECK(hipGetDevice(&current_device));

    if (device == current_device) {
        return hipSuccess;
    }

    return hipSetDevice(device);
}



void ggml_init_cublas() {
    static bool initialized = false;

    if (!initialized) {

#ifdef __HIP_PLATFORM_AMD__
        // Workaround for a rocBLAS bug when using multiple graphics cards:
        // https://github.com/ROCmSoftwarePlatform/rocBLAS/issues/1346
        rocblas_initialize();
        CUDA_CHECK(hipDeviceSynchronize());
#endif

        if (hipGetDeviceCount(&g_device_count) != hipSuccess) {
            initialized = true;
            g_cublas_loaded = false;
            return;
        }

        GGML_ASSERT(g_device_count <= GGML_CUDA_MAX_DEVICES);
        int64_t total_vram = 0;
#if defined(GGML_CUDA_FORCE_MMQ)
        fprintf(stderr, "%s: GGML_CUDA_FORCE_MMQ:   yes\n", __func__);
#else
        fprintf(stderr, "%s: GGML_CUDA_FORCE_MMQ:   no\n", __func__);
#endif
#if defined(CUDA_USE_TENSOR_CORES)
        fprintf(stderr, "%s: CUDA_USE_TENSOR_CORES: yes\n", __func__);
#else
        fprintf(stderr, "%s: CUDA_USE_TENSOR_CORES: no\n", __func__);
#endif
        fprintf(stderr, "%s: found %d " GGML_CUDA_NAME " devices:\n", __func__, g_device_count);
        for (int id = 0; id < g_device_count; ++id) {
            hipDeviceProp_t prop;
            CUDA_CHECK(hipGetDeviceProperties(&prop, id));
            fprintf(stderr, "  Device %d: %s, compute capability %d.%d\n", id, prop.name, prop.major, prop.minor);

            g_tensor_split[id] = total_vram;
            total_vram += prop.totalGlobalMem;
#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
            g_compute_capabilities[id] = 100*prop.major + 10*prop.minor + CC_OFFSET_AMD;
#else
            g_compute_capabilities[id] = 100*prop.major + 10*prop.minor;
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)
        }
        for (int id = 0; id < g_device_count; ++id) {
            g_tensor_split[id] /= total_vram;
        }

        for (int id = 0; id < g_device_count; ++id) {
            CUDA_CHECK(ggml_cuda_set_device(id));

            // create cuda streams
            for (int is = 0; is < MAX_STREAMS; ++is) {
                CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams[id][is], hipStreamNonBlocking));
            }

            // create cublas handle
            CUBLAS_CHECK(hipblasCreate(&g_cublas_handles[id]));
            CUBLAS_CHECK(hipblasSetMathMode(g_cublas_handles[id], HIPBLAS_TF32_TENSOR_OP_MATH));
        }

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, nullptr));

        initialized = true;
        g_cublas_loaded = true;
    }
}


void * ggml_cuda_host_malloc(size_t size) {
    if (getenv("GGML_CUDA_NO_PINNED") != nullptr) {
        return nullptr;
    }

    void * ptr = nullptr;
    hipError_t err = hipHostMalloc((void **) &ptr, size);
    if (err != hipSuccess) {
        // The allocation error can be bypassed. A null ptr will assigned out of this function.
        // This can fixed the OOM error in WSL.
        hipGetLastError();
        fprintf(stderr, "WARNING: failed to allocate %.2f MB of pinned memory: %s\n",
            size/1024.0/1024.0, hipGetErrorString(err));
        return nullptr;
    }

    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

